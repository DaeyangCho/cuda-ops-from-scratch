#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < M && c < K) {
        float v = 0.0;
        for (int i=0; i<N; i++) {
            v += A[(r * N) + i] * B[(K * i) + c];
        }
        C[(r * K) + c] = v;
    }
    
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
