#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cmath>
#include <cstdio>
#include <random>
#include <vector>
#include "../utils/timer.h"

// Global memory coalesced matrix multiplication kernel
// Traverses consecutive columns of matrix B so that threads in a warp access
// contiguous memory locations, enabling global memory coalescing
__global__ void matrix_multiplication_kernel(const float* A, const float* B,
                                             float* C, int M, int N, int K) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < N && y < M) {
        float v = 0.0;
        for (int i = 0; i < K; i++) {
            v += A[(y * K) + i] * B[(i * N) + x];
        }
        C[(y * N) + x] = v;
    }
}

int main() {
    // Matrix dimensions: A(M×K), B(K×N), C(M×N)
    const int M = 1024;
    const int N = 1024;
    const int K = 1024;

    // Host init
    std::vector<float> ha(M * K), hb(K * N), hc(M * N);
    std::mt19937 rng(42);
    std::uniform_real_distribution<float> dist(0.f, 1.f);
    for (int i = 0; i < M * K; ++i) ha[i] = dist(rng);
    for (int i = 0; i < K * N; ++i) hb[i] = dist(rng);

    // Device alloc
    float *da, *db, *dc;
    CUDA_CHECK(hipMalloc(&da, M * K * sizeof(float)));
    CUDA_CHECK(hipMalloc(&db, K * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&dc, M * N * sizeof(float)));

    CUDA_CHECK(hipMemcpy(da, ha.data(), M * K * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(db, hb.data(), K * N * sizeof(float),
                          hipMemcpyHostToDevice));

    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    // Warm-up
    matrix_multiplication_kernel<<<grid, block>>>(da, db, dc, M, N, K);
    CUDA_CHECK(hipDeviceSynchronize());

    CudaTimer t;
    t.start();
    for (int r = 0; r < 100; r++)
        matrix_multiplication_kernel<<<grid, block>>>(da, db, dc, M, N, K);
    CUDA_CHECK(hipDeviceSynchronize());
    float ms = t.stop_ms() / 100.0f;

    CUDA_CHECK(hipMemcpy(hc.data(), dc, M * N * sizeof(float),
                          hipMemcpyDeviceToHost));

    // Verify
    double max_err = 0.0;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float expected = 0.0f;
            for (int k = 0; k < K; ++k) {
                expected += ha[i * M + k] * hb[k * N + j];
            }
            max_err = std::max(
                max_err,
                static_cast<double>(std::fabs(hc[i * N + j] - expected)));
        }
    }

    printf(
        "cuda 2d gmem matrix_multiply: M=%d, N=%d, K=%d, time=%.4f ms, "
        "max_err=%.3g\n",
        M, N, K, ms, max_err);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}