#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdio>
#include <random>
#include <vector>
#include "../utils/timer.h"

__global__ void vector_add_kernel(const float *__restrict__ a,
                                  const float *__restrict__ b,
                                  float *__restrict__ c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    const int N = 200'000'000;
    const size_t bytes = N * sizeof(float);

    // Host init
    std::vector<float> ha(N), hb(N), hc(N);
    std::mt19937 rng(42);
    std::uniform_real_distribution<float> dist(0.f, 1.f);
    for (int i = 0; i < N; ++i) {
        ha[i] = dist(rng);
        hb[i] = dist(rng);
    }

    // Device alloc
    float *da, *db, *dc;
    CUDA_CHECK(hipMalloc(&da, bytes));
    CUDA_CHECK(hipMalloc(&db, bytes));
    CUDA_CHECK(hipMalloc(&dc, bytes));

    CUDA_CHECK(hipMemcpy(da, ha.data(), bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(db, hb.data(), bytes, hipMemcpyHostToDevice));

    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);

    // Warm-up
    vector_add_kernel<<<grid, block>>>(da, db, dc, N);
    CUDA_CHECK(hipDeviceSynchronize());

    CudaTimer t;
    t.start();
    for (int r = 0; r < 100; r++)
        vector_add_kernel<<<grid, block>>>(da, db, dc, N);
    CUDA_CHECK(hipDeviceSynchronize());
    float ms = t.stop_ms() / 100.0f;

    CUDA_CHECK(hipMemcpy(hc.data(), dc, bytes, hipMemcpyDeviceToHost));

    // Verify
    double max_err = 0.0;
    for (int i = 0; i < N; ++i)
        max_err = std::max(max_err, (double)std::abs(hc[i] - (ha[i] + hb[i])));

    printf("vector_add: N=%d, time=%.4f ms, max_err=%.3g\n", N, ms, max_err);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}